#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/batch_renorm_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

	template <typename Dtype>
	__global__ void R_D_CUT(const int n, Dtype* r, Dtype* d
		, Dtype cur_r_max, Dtype cur_r_min, Dtype cur_d_max, Dtype cur_d_min) {
		CUDA_KERNEL_LOOP(index, n) {
			r[index] = __min(cur_r_max, __max(r[index], cur_r_min));
			d[index] = __min(cur_d_max, __max(d[index], cur_d_min));
		}
	}

	template <typename Dtype>
	void BatchReNormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();
		int num = bottom[0]->shape(0);
		int spatial_dim = bottom[0]->count() / (channels_*bottom[0]->shape(0));
		Dtype iter;
    caffe_copy(1, this->blobs_[3]->gpu_data(), &iter);
		int step = int(iter) / iter_size_;
		bool first_iter_in_step = (int(iter)%iter_size_ == 0);

    //LOG(INFO) << this->layer_param_.name() << " iter:" << iter << ", step:" << step;

    /* Use top[1], top[2], top[3], and top[4] for temp, x_norm, all_r, and all_d */
    Blob<Dtype>& temp = *top[1];
    Blob<Dtype>& x_norm = *top[2];
    Blob<Dtype>& all_r = *top[3];
    Blob<Dtype>& all_d = *top[4];
    

		if (bottom[0] != top[0]) {
			caffe_copy(bottom[0]->count(), bottom_data, top_data);
		}

		if (use_global_stats_) {
			// use the stored mean/variance estimates.
      Dtype m_counter; 
      caffe_copy(1, this->blobs_[2]->gpu_data(), &m_counter);
      const Dtype scale_factor = m_counter == 0 ? 0 : 1 / m_counter;
			caffe_gpu_scale(variance_.count(), scale_factor,
				this->blobs_[0]->gpu_data(), mean_.mutable_gpu_data());
			caffe_gpu_scale(variance_.count(), scale_factor,
				this->blobs_[1]->gpu_data(), variance_.mutable_gpu_data());
      if (0 == Caffe::worker_id()) {
        LOG(INFO) << this->layer_param_.name() << " iter:" << iter << ", m_counter:" << m_counter;
      }
		}
		else {
			// compute mean
			caffe_gpu_gemv<Dtype>(CblasNoTrans, channels_ * num, spatial_dim,
				1. / (num * spatial_dim), bottom_data,
				spatial_sum_multiplier_.gpu_data(), 0.,
				num_by_chans_.mutable_gpu_data());
			caffe_gpu_gemv<Dtype>(CblasTrans, num, channels_, 1.,
				num_by_chans_.gpu_data(), batch_sum_multiplier_.gpu_data(), 0.,
				mean_.mutable_gpu_data());
		}

		// subtract mean
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
			batch_sum_multiplier_.gpu_data(), mean_.gpu_data(), 0.,
			num_by_chans_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
			spatial_dim, 1, -1, num_by_chans_.gpu_data(),
			spatial_sum_multiplier_.gpu_data(), 1., top_data);

		if (!use_global_stats_) {
			// compute variance using var(X) = E((X-EX)^2)
			caffe_gpu_powx(top[0]->count(), top_data, Dtype(2),
				temp.mutable_gpu_data());  // (X-EX)^2
			caffe_gpu_gemv<Dtype>(CblasNoTrans, channels_ * num, spatial_dim,
				1. / (num * spatial_dim), temp.gpu_data(),
				spatial_sum_multiplier_.gpu_data(), 0.,
				num_by_chans_.mutable_gpu_data());
			caffe_gpu_gemv<Dtype>(CblasTrans, num, channels_, 1.,
				num_by_chans_.gpu_data(), batch_sum_multiplier_.gpu_data(), 0.,
				variance_.mutable_gpu_data());  // E((X_EX)^2)

			if (step >= step_to_init_ && first_iter_in_step)
			{
        Dtype m_counter; 
        caffe_copy(1, this->blobs_[2]->gpu_data(), &m_counter);
				const Dtype scale_factor = 1. / m_counter;
				caffe_gpu_scale(variance_.count(), scale_factor, this->blobs_[0]->gpu_data(), 
                        mean_glb_.mutable_gpu_data());
				caffe_gpu_scale(variance_.count(), scale_factor, this->blobs_[1]->gpu_data(), 
                        variance_glb_.mutable_gpu_data());
				caffe_gpu_add_scalar(variance_.count(), eps_, variance_glb_.mutable_gpu_data());
				caffe_gpu_powx(variance_.count(), this->variance_glb_.gpu_data(), Dtype(0.5), 
                       this->variance_glb_.mutable_gpu_data());
			}

      // Backup the variance here so we can calculate moving average at
      // the backward phase
      caffe_copy(variance_.count(), variance_.gpu_data(), 
                 variance_back_.mutable_gpu_data());
			// compute and save moving average
			//Dtype moving_average_fraction = first_iter_in_step ? moving_average_fraction_ : 1.0;
			//this->blobs_[2]->mutable_cpu_data()[0] *= moving_average_fraction;
			//this->blobs_[2]->mutable_cpu_data()[0] += 1;
			//caffe_gpu_axpby(mean_.count(), Dtype(1), mean_.gpu_data(),
			//	moving_average_fraction, this->blobs_[0]->mutable_gpu_data());
			//int m = bottom[0]->count() / channels_;
			//Dtype bias_correction_factor = m > 1 ? Dtype(m) / (m - 1) : 1;
			//caffe_gpu_axpby(variance_.count(), bias_correction_factor,
			//	variance_.gpu_data(), moving_average_fraction,
			//	this->blobs_[1]->mutable_gpu_data());
		}

		// normalize variance
		caffe_gpu_add_scalar(variance_.count(), eps_, variance_.mutable_gpu_data());
		caffe_gpu_powx(variance_.count(), variance_.gpu_data(), Dtype(0.5),
			variance_.mutable_gpu_data());

		// replicate variance to input size
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
			batch_sum_multiplier_.gpu_data(), variance_.gpu_data(), 0.,
			num_by_chans_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
			spatial_dim, 1, 1., num_by_chans_.gpu_data(),
			spatial_sum_multiplier_.gpu_data(), 0., temp.mutable_gpu_data());
		caffe_gpu_div(temp.count(), top_data, temp.gpu_data(), top_data);
		// TODO(cdoersch): The caching is only needed because later in-place layers
		//                 might clobber the data.  Can we skip this if they won't?
		caffe_copy(x_norm.count(), top_data,
			x_norm.mutable_gpu_data());

		if (!use_global_stats_ && step >= step_to_init_)
		{
			Dtype cur_r_max = __max(1, __min(1 + (step - step_to_init_ + 1)*(r_max_ - 1) / (step_to_r_max_ - step_to_init_), r_max_));
			Dtype cur_r_min = 1. / cur_r_max;
			Dtype cur_d_max = __max(0, __min((step - step_to_init_ + 1)*d_max_ / (step_to_d_max_ - step_to_init_), d_max_));
			Dtype cur_d_min = -cur_d_max;

			caffe_gpu_div(variance_.count(), variance_.gpu_data(), variance_glb_.gpu_data(), 
                    r_.mutable_gpu_data());

			caffe_copy(variance_.count(), mean_.gpu_data(), d_.mutable_gpu_data());
			caffe_gpu_axpby(variance_.count(), Dtype(-1), mean_glb_.gpu_data(), 
                      Dtype(1), d_.mutable_gpu_data());
			caffe_gpu_div(variance_.count(), d_.gpu_data(), variance_glb_.gpu_data(), 
                    d_.mutable_gpu_data());

			R_D_CUT<Dtype> << <CAFFE_GET_BLOCKS(variance_.count()), CAFFE_CUDA_NUM_THREADS >> >(
				variance_.count(), r_.mutable_gpu_data(), d_.mutable_gpu_data(), cur_r_max, cur_r_min
				, cur_d_max, cur_d_min);
			CUDA_POST_KERNEL_CHECK;

			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
				batch_sum_multiplier_.gpu_data(), r_.gpu_data(), 0.,
				num_by_chans_.mutable_gpu_data());
			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
				spatial_dim, 1, 1., num_by_chans_.gpu_data(),
				spatial_sum_multiplier_.gpu_data(), 0., all_r.mutable_gpu_data());
			caffe_gpu_mul(temp.count(), top_data, all_r.gpu_data(), top_data);

			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
				batch_sum_multiplier_.gpu_data(), d_.gpu_data(), 0.,
				num_by_chans_.mutable_gpu_data());
			caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
				spatial_dim, 1, 1., num_by_chans_.gpu_data(),
				spatial_sum_multiplier_.gpu_data(), 0., all_d.mutable_gpu_data());
			caffe_gpu_add(temp.count(), top_data, all_d.gpu_data(), top_data);
		}
	}

	template <typename Dtype>
	void BatchReNormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom) {
		const Dtype* top_diff;

    /* Use top[1], top[2], top[3], and top[4] for temp, x_norm, all_r, and all_d */
    Blob<Dtype>& temp = *top[1];
    Blob<Dtype>& x_norm = *top[2];
    Blob<Dtype>& all_r = *top[3];
    Blob<Dtype>& all_d = *top[4];

		Dtype iter;
    caffe_copy(1, this->blobs_[3]->gpu_data(), &iter);
		int step = int(iter) / iter_size_;

		if (bottom[0] != top[0]) {
			top_diff = top[0]->gpu_diff();
		}
		else {
			caffe_copy(x_norm.count(), top[0]->gpu_diff(), x_norm.mutable_gpu_diff());
			top_diff = x_norm.gpu_diff();
		}
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

		if (use_global_stats_)
		{
			caffe_gpu_div(temp.count(), top_diff, temp.gpu_data(), bottom_diff);
			return;
		}

    // compute and save moving average
    if (!use_global_stats_) {
      const Dtype* m_counter = this->blobs_[2]->gpu_data();    
      Dtype* m_counter_diff = this->blobs_[2]->mutable_gpu_diff();    

      const Dtype* m_average = this->blobs_[0]->gpu_data();
      Dtype* m_average_diff = this->blobs_[0]->mutable_gpu_diff();    

      const Dtype* m_variance = this->blobs_[1]->gpu_data();
      Dtype* m_variance_diff = this->blobs_[1]->mutable_gpu_diff();
      int m = bottom[0]->count()/channels_;
      Dtype bias_correction_factor = m > 1 ? Dtype(m)/(m-1) : 1;

      // We only apply moving average fraction on worker 0
      //if (0 == Caffe::worker_id()) {
      {
        caffe_gpu_axpy(this->blobs_[2]->count(), moving_average_fraction_ - Dtype(1),
                       m_counter, m_counter_diff);
        caffe_gpu_axpy(mean_.count(), moving_average_fraction_ - Dtype(1), 
                       m_average, m_average_diff);
        caffe_gpu_axpy(variance_back_.count(), moving_average_fraction_ - Dtype(1), 
                       m_variance, m_variance_diff);
      }

      caffe_gpu_add_scalar(this->blobs_[2]->count(), Dtype(1), m_counter_diff);
      caffe_gpu_axpy(mean_.count(), Dtype(1), mean_.gpu_data(), m_average_diff);
      caffe_gpu_axpy(variance_back_.count(), bias_correction_factor, 
                     variance_back_.gpu_data(), m_variance_diff);
    }

		const Dtype* top_data = x_norm.gpu_data();
		int num = bottom[0]->shape()[0];
		int spatial_dim = bottom[0]->count() / (channels_*bottom[0]->shape(0));
		// if Y = (X-mean(X))/(sqrt(var(X)+eps)), then
		//
		// dE(Y)/dX =
		//   (dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y)
		//     ./ sqrt(var(X) + eps)
		//
		// where \cdot and ./ are hadamard product and elementwise division,
		// respectively, dE/dY is the top diff, and mean/var/sum are all computed
		// along all dimensions except the channels dimension.  In the above
		// equation, the operations allow for expansion (i.e. broadcast) along all
		// dimensions except the channels dimension where required.

		// sum(dE/dY \cdot Y)
		caffe_gpu_mul(temp.count(), top_data, top_diff, bottom_diff);
		caffe_gpu_gemv<Dtype>(CblasNoTrans, channels_ * num, spatial_dim, 1.,
			bottom_diff, spatial_sum_multiplier_.gpu_data(), 0.,
			num_by_chans_.mutable_gpu_data());
		caffe_gpu_gemv<Dtype>(CblasTrans, num, channels_, 1.,
			num_by_chans_.gpu_data(), batch_sum_multiplier_.gpu_data(), 0.,
			mean_.mutable_gpu_data());

		// reshape (broadcast) the above
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
			batch_sum_multiplier_.gpu_data(), mean_.gpu_data(), 0.,
			num_by_chans_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
			spatial_dim, 1, 1., num_by_chans_.gpu_data(),
			spatial_sum_multiplier_.gpu_data(), 0., bottom_diff);

		// sum(dE/dY \cdot Y) \cdot Y
		caffe_gpu_mul(temp.count(), top_data, bottom_diff, bottom_diff);

		// sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
		caffe_gpu_gemv<Dtype>(CblasNoTrans, channels_ * num, spatial_dim, 1.,
			top_diff, spatial_sum_multiplier_.gpu_data(), 0.,
			num_by_chans_.mutable_gpu_data());
		caffe_gpu_gemv<Dtype>(CblasTrans, num, channels_, 1.,
			num_by_chans_.gpu_data(), batch_sum_multiplier_.gpu_data(), 0.,
			mean_.mutable_gpu_data());
		// reshape (broadcast) the above to make
		// sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
			batch_sum_multiplier_.gpu_data(), mean_.gpu_data(), 0.,
			num_by_chans_.mutable_gpu_data());
		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num * channels_,
			spatial_dim, 1, 1., num_by_chans_.gpu_data(),
			spatial_sum_multiplier_.gpu_data(), 1., bottom_diff);

		// dE/dY - mean(dE/dY)-mean(dE/dY \cdot Y) \cdot Y
		caffe_gpu_axpby(temp.count(), Dtype(1), top_diff,
			Dtype(-1. / (num * spatial_dim)), bottom_diff);

		// note: temp still contains sqrt(var(X)+eps), computed during the forward
		// pass.
		caffe_gpu_div(temp.count(), bottom_diff, temp.gpu_data(), bottom_diff);
		

		if (!use_global_stats_ && step >= step_to_init_)
		{
			caffe_gpu_mul(temp.count(), bottom_diff, all_r.gpu_data(), bottom_diff);
		}

		if (this->phase_ == TRAIN && 0 == Caffe::worker_id())
    {
      caffe_gpu_add_scalar(this->blobs_[3]->count(), Dtype(1), 
                           this->blobs_[3]->mutable_gpu_diff());
    }
	}

	INSTANTIATE_LAYER_GPU_FUNCS(BatchReNormLayer);


}  // namespace caffe
